/*
 * SPDX-License-Identifier: Apache-2.0
 * Copyright(c) 2024 Liu, Changcheng <changcheng.liu@aliyun.com>
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

/* $ nvcc -gencode arch=compute_89,code=sm_89 -o matrixAdd matrixAdd.cu
 * $ ./matrixAdd
 *   Resulting Matrix C:
 *   0.000000 0.000000 0.000000 0.000000
 *   2.000000 2.000000 2.000000 2.000000
 *   4.000000 4.000000 4.000000 4.000000
 *   6.000000 6.000000 6.000000 6.000000
 */

#define N 4 // Define the size of the matrices

__global__
void MatAdd(float A[N][N], float B[N][N], float C[N][N])
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < N) {
        C[i][j] = A[i][j] + B[i][j];
    }
}

int main() {
    size_t size = N * N * sizeof(float);
    
    // Allocate host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize input matrices
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
           /* h_A {{0,  1,  2,  3},
            *      {1,  2,  3,  4},
            *      {2,  3,  4,  5},
            *      {3,  4,  5,  6}}
            */
            h_A[i * N + j] = i + j;

           /* h_B {{0, -1, -2, -3},
            *      {1,  0, -1, -2},
            *      {2,  1,  0, -1},
            *      {3,  2,  1,  0}
            */
            h_B[i * N + j] = i - j;
        }
    }

    // Allocate device memory
    float (*d_A)[N], (*d_B)[N], (*d_C)[N];
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Kernel invocation
    dim3 threadsPerBlock(2, 2);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    MatAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print the result
    printf("Resulting Matrix C:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%08f ", h_C[i * N + j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
