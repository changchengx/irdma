/*
 * SPDX-License-Identifier: Apache-2.0
 * Copyright(c) 2024 Liu, Changcheng <changcheng.liu@aliyun.com>
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

/*Build: take L40S for example
 * $ nvidia-smi -L
 * GPU 0: NVIDIA L40S (UUID: GPU-b5efa9ba-8fda-999e-42e9-409f10cb6f78)
 * GPU 1: NVIDIA L40S (UUID: GPU-1ee92b64-2d96-843e-557b-4331a8004649)
 * GPU 2: NVIDIA L40S (UUID: GPU-29f96d36-7a69-22c5-ce1d-fd4d76f698de)
 * GPU 3: NVIDIA L40S (UUID: GPU-bf1d4cd9-0d39-7e27-de3b-0ca9815a26da)
 *
 * $ nvidia-smi --format=csv --query-gpu=compute_cap -i GPU-b5efa9ba-8fda-999e-42e9-409f10cb6f78
 * compute_cap
 * 8.9
 *
 * $ nvcc -gencode arch=compute_89,code=sm_89 -o vectorAdd vectorAdd.cu
 *
 *Run:
 * $ ./vectorAdd
 * Result: 2.000000 4.000000 6.000000 8.000000
 */

__global__
void VecAdd(float* A, float* B, float* C, int N)
{
    int i = threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 4;
    size_t size = N * sizeof(float);

    // Allocate input vectors h_A and h_B in host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize input vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = i + 1; // A = {1, 2, 3, 4}
        h_B[i] = i + 1; // B = {1, 2, 3, 4}
    }

    // Allocate vectors in device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel
    VecAdd<<<1, N>>>(d_A, d_B, d_C, N);

    // Copy result from device memory to host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print the result
    printf("Result: ");
    for (int i = 0; i < N; i++) {
        printf("%f ", h_C[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
